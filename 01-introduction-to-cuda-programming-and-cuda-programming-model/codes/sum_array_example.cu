#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void sum_array_gpu(int *a, int *b, int *c, int size)
{

  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size)
  {
    c[gid] = a[gid] + b[gid];
  }
}
void sum_array_cpu(int *a, int *b, int *c, int size)
{
  for (int i = 0; i < size; i++)
  {
    c[i] = a[i] + b[i];
  }
}
void compare_arrays(int *a, int *b, int size)
{ // compare arrays
  for (int i = 0; i < size; i++)
  {
    if (a[i] != b[i])
    {
      printf("Arrays are different \n");
      return;
    }
  }
  printf("Arrays are the same \n");
}

int main()
{

  int size = 10000;
  int block_size = 128;

  int NO_BYTES = size * sizeof(int);

  int *h_a, *h_b, *gpu_results, *h_c;

  h_a = (int *)malloc(NO_BYTES);
  h_b = (int *)malloc(NO_BYTES);
  h_c = (int *)malloc(NO_BYTES);
  gpu_results = (int *)malloc(NO_BYTES);

  time_t t;
  srand((unsigned)time(&t));
  for (int i = 0; i < size; i++)
  {
    h_a[i] = (int)(rand() & 0xff);
  }
  for (int i = 0; i < size; i++)
  {
    h_b[i] = (int)(rand() & 0xff);
  }
  sum_array_cpu(h_a, h_b, h_c, size);

  // device pointer

  int *d_a, *d_b, *d_c;
  hipMalloc((int **)&d_a, NO_BYTES);
  hipMalloc((int **)&d_b, NO_BYTES);
  hipMalloc((int **)&d_c, NO_BYTES);

  hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

  dim3 block(block_size);
  dim3 grid((size / block.x) + 1);

  sum_array_gpu<<<grid, block>>>(d_a, d_b, d_c, size);
  hipDeviceSynchronize();
  hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);

  compare_arrays(gpu_results, h_c, size);

  hipFree(d_c);
  hipFree(d_b);
  hipFree(d_a);

  free(gpu_results);
  free(h_a);
  free(h_b);
  free(h_c);

  hipDeviceReset();
  return 0;
}