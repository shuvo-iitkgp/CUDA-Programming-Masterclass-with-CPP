#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda()
{
  printf("Hello CUDA WORLD \n");
}

int main()
{
  hello_cuda<<<20, 1>>>();
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}