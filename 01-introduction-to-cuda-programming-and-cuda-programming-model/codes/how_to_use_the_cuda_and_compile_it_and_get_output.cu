
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void helloFromGPU()
{
    printf("Hello from the GPU thread!\n");
}

int main()
{
    // Launching the kernel with 1 block and 1 thread
    helloFromGPU<<<1, 1>>>();

    // Checking if the kernel launched properly
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Synchronize the device to make sure the output is flushed
    hipDeviceSynchronize();

    return 0;
}
