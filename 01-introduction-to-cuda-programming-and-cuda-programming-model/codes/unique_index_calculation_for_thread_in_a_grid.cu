#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void unique_idx_calc_threadIdx(int *input)
{
  int tid = threadIdx.x;
  printf("threadIdx: %d, value : %d \n ", tid, input[tid]);
}

int main()
{

  int array_size = 8;
  int array_byte_size = array_size * sizeof(int);
  int h_data[] = {24, 4, 23, 42, 1, 4, 2, 3};
  for (int i = 0; i < array_size; i++)
  {
    printf("%d ", h_data[i]);
  }
  printf("\n\n");
  int *d_data;
  hipMalloc((void **)&d_data, array_byte_size);
  hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

  dim3 block(8);
  dim3 grid(1);

  unique_idx_calc_threadIdx<<<grid, block>>>(d_data);

  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}